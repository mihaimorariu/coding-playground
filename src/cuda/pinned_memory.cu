#include "common/book.h"

#define SIZE (10 * 1024 * 1024)

float cuda_host_alloc_test(int size, bool up) {
  hipEvent_t start, stop;
  int *a, *dev_a;
  float elapsed_time;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  HANDLE_ERROR(hipHostAlloc((void**)&a, size * sizeof(*a), hipHostMallocDefault));
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(*dev_a)));
  HANDLE_ERROR(hipEventRecord(start, 0));

  for (int i = 0; i < 100; ++i) {
    if (up) {
      HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice));
    } else {
      HANDLE_ERROR(hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost));
    }
  }

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time, start, stop));

  HANDLE_ERROR(hipHostFree(a));
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  return elapsed_time;
}

float cuda_malloc_test(int size, bool up) {
  hipEvent_t start, stop;
  int *a, *dev_a;
  float elapsed_time;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  a = (int*)malloc(size * sizeof(a));
  HANDLE_NULL(a);
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(*dev_a)));
  HANDLE_ERROR(hipEventRecord(start, 0));

  for (int i = 0; i < 100; ++i) {
    if (up) {
      HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice));
    } else {
      HANDLE_ERROR(hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost));
    }
  }

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time, start, stop));

  free(a);
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  return elapsed_time;
}

int main(int argc, char *argv[]) {
  float elapsed_time;
  float MB = (float)100 * SIZE * sizeof(int) / 1024 / 1024;

  elapsed_time = cuda_malloc_test(SIZE, true);
  printf("Time using hipMalloc: %3.1f ms\n", elapsed_time);
  printf("\tMB/s during copy up: %3.1f\n", MB / (elapsed_time / 1000));

  elapsed_time = cuda_malloc_test(SIZE, false);
  printf("Time using hipMalloc: %3.1f ms\n", elapsed_time);
  printf("\tMB/s during copy down: %3.1f\n", MB / (elapsed_time / 1000));

  elapsed_time = cuda_host_alloc_test(SIZE, true);
  printf("Time using hipHostAlloc: %3.1f ms\n", elapsed_time);
  printf("\tMB/s during copy down: %3.1f\n", MB / (elapsed_time / 1000));

  elapsed_time = cuda_host_alloc_test(SIZE, false);
  printf("Time using hipHostAlloc: %3.1f ms\n", elapsed_time);
  printf("\tMB/s during copy down: %3.1f\n", MB / (elapsed_time / 1000));

  return EXIT_SUCCESS;
}
