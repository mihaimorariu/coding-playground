#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f

texture<float, 2> tex_const_src;
texture<float, 2> tex_in;
texture<float, 2> tex_out;

__global__ void copy_const_kernel(float *iptr) {
  int x      = threadIdx.x + blockIdx.x * blockDim.x;
  int y      = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  float c = tex2D(tex_const_src, x, y);
  if (c != 0) { iptr[offset] = c; }
}

__global__ void blend_kernel(float *dst, bool dst_out) {
  int x      = threadIdx.x + blockIdx.x * blockDim.x;
  int y      = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  float t, l, c, r, b;
  if (dst_out) {
    t = tex2D(tex_in, x, y - 1);
    l = tex2D(tex_in, x - 1, y);
    c = tex2D(tex_in, x, y);
    r = tex2D(tex_in, x + 1, y);
    b = tex2D(tex_in, x, y + 1);
  } else {
    t = tex2D(tex_out, x, y - 1);
    l = tex2D(tex_out, x - 1, y);
    c = tex2D(tex_out, x, y);
    r = tex2D(tex_out, x + 1, y);
    b = tex2D(tex_out, x, y + 1);
  }

  dst[offset] = c + SPEED * (t + b + l + r - 4 * c);
}

struct DataBlock {
  unsigned char *output_bitmap;
  float         *dev_in_src;
  float         *dev_out_src;
  float         *dev_const_src;
  CPUAnimBitmap *bitmap;
  hipEvent_t    start, stop;
  float          total_time;
  float          frames;
};

void anim_gpu(DataBlock *d, int ticks) {
  HANDLE_ERROR(hipEventRecord(d->start, 0));
  dim3 blocks(DIM / 16, DIM / 16);
  dim3 threads(16, 16);
  CPUAnimBitmap *bitmap = d->bitmap;

  volatile bool dst_out = true;
  for (int i = 0; i < 90; ++i) {
    float *in  = dst_out ? d->dev_in_src  : d->dev_out_src;
    float *out = dst_out ? d->dev_out_src : d->dev_in_src;

    copy_const_kernel<<<blocks, threads>>>(in);
    blend_kernel<<<blocks, threads>>>(out, dst_out);
    dst_out = !dst_out;
  }

  float_to_color<<<blocks, threads>>>(d->output_bitmap, d->dev_in_src);

  HANDLE_ERROR(hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(),
                          hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipEventRecord(d->stop, 0));
  HANDLE_ERROR(hipEventSynchronize(d->stop));

  float elapsed_time;
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time, d->start, d->stop));

  d->total_time += elapsed_time;
  ++d->frames;
  printf("Average time per frame: %3.1f ms\n", d->total_time / d->frames);
}

void anim_exit(DataBlock *d) {
  hipUnbindTexture(tex_in);
  hipUnbindTexture(tex_out);
  hipUnbindTexture(tex_const_src);

  hipFree(d->dev_in_src);
  hipFree(d->dev_out_src);
  hipFree(d->dev_const_src);

  HANDLE_ERROR(hipEventDestroy(d->start));
  HANDLE_ERROR(hipEventDestroy(d->stop));
}

int main(int argc, char *argv[]) {
  DataBlock data;
  CPUAnimBitmap bitmap(DIM, DIM, &data);

  data.bitmap     = &bitmap;
  data.total_time = 0;
  data.frames     = 0;

  HANDLE_ERROR(hipEventCreate(&data.start));
  HANDLE_ERROR(hipEventCreate(&data.stop));

  HANDLE_ERROR(hipMalloc((void**)&data.output_bitmap, bitmap.image_size()));

  HANDLE_ERROR(hipMalloc((void**)&data.dev_in_src,    bitmap.image_size()));
  HANDLE_ERROR(hipMalloc((void**)&data.dev_out_src,   bitmap.image_size()));
  HANDLE_ERROR(hipMalloc((void**)&data.dev_const_src, bitmap.image_size()));

  hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
  HANDLE_ERROR(hipBindTexture2D(NULL, tex_const_src, data.dev_const_src, desc, DIM, DIM, sizeof(float) * DIM));
  HANDLE_ERROR(hipBindTexture2D(NULL, tex_in,        data.dev_in_src,    desc, DIM, DIM, sizeof(float) * DIM));
  HANDLE_ERROR(hipBindTexture2D(NULL, tex_out,       data.dev_out_src,   desc, DIM, DIM, sizeof(float) * DIM));

  float *temp = (float*)malloc(bitmap.image_size());
  for (int i = 0; i < DIM * DIM; ++i) {
    temp[i] = 0;

    int x = i % DIM;
    int y = i / DIM;

    if (x > 300 && x < 600 && y > 310 && y < 601) { temp[i] = MAX_TEMP; }
  }

  temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
  temp[DIM * 700 + 100] = MIN_TEMP;
  temp[DIM * 300 + 300] = MIN_TEMP;
  temp[DIM * 200 + 700] = MIN_TEMP;

  for (int y = 800; y < 900; ++y) {
    for (int x = 400; x < 500; ++x) {
      temp[x + y * DIM] = MIN_TEMP;
    }
  }

  HANDLE_ERROR(hipMemcpy(data.dev_const_src, temp, bitmap.image_size(), hipMemcpyHostToDevice));

  for (int y = 800; y < DIM; ++y) {
    for (int x = 0; x < 200; ++x) {
      temp[x + y * DIM] = MAX_TEMP;
    }
  }

  HANDLE_ERROR(hipMemcpy(data.dev_in_src, temp, bitmap.image_size(), hipMemcpyHostToDevice));

  free(temp);
  bitmap.anim_and_exit((void(*)(void*, int))anim_gpu, (void(*)(void*))anim_exit);

  return EXIT_SUCCESS;
}

