#include "hip/hip_runtime.h"
#include "common/book.h"

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

#define N 10

__global__ void add(int *a, int *b, int *c) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    c[tid] = a[tid] + b[tid];
    tid += blockDim.x * gridDim.x;
  }
}

int main(void) {
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  HANDLE_ERROR(hipMalloc((void **)&dev_a, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_c, N * sizeof(int)));

  for (int i = 0; i < N; ++i) {
    a[i] = -i;
    b[i] = i * i;
  }

  HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

  add<<<128, 128>>>(dev_a, dev_b, dev_c);

  HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

  for (int i = 0; i < N; ++i) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}
