#include "common/book.h"

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

int main(void) {
  hipDeviceProp_t prop;
  int dev;

  HANDLE_ERROR(hipGetDevice(&dev));

  memset(&prop, 0, sizeof(hipDeviceProp_t));
  prop.major = 1;
  prop.minor = 3;

  HANDLE_ERROR(hipChooseDevice(&dev, &prop));

  printf("ID of CUDA device closest to revision 1.3: %d\n", dev);

  HANDLE_ERROR(hipSetDevice(dev));

  return 0;
}
