#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/cpu_anim.h"

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f

texture<float> tex_const_src;
texture<float> tex_in;
texture<float> tex_out;

struct DataBlock {
  unsigned char *output_bitmap;

  float *dev_in_src;
  float *dev_out_src;
  float *dev_const_src;

  CPUAnimBitmap *bitmap;
  hipEvent_t start, stop;

  float total_time;
  float frames;
};

__global__ void copy_const_kernel(float *iptr) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  float c = tex1Dfetch(tex_const_src, offset);

  if (c != 0) {
    iptr[offset] = c;
  }
}

__global__ void blend_kernel(float *dst, bool dst_out) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  int left = offset - 1;
  int right = offset + 1;

  if (x == 0) {
    ++left;
  }

  if (x == DIM - 1) {
    --right;
  }

  int top = offset - DIM;
  int bottom = offset + DIM;

  if (y == 0) {
    top += DIM;
  }

  if (y == DIM - 1) {
    bottom -= DIM;
  }

  float t, l, c, r, b;

  if (dst_out) {
    t = tex1Dfetch(tex_in, top);
    l = tex1Dfetch(tex_in, left);
    c = tex1Dfetch(tex_in, offset);
    r = tex1Dfetch(tex_in, right);
    b = tex1Dfetch(tex_in, bottom);
  } else {
    t = tex1Dfetch(tex_out, top);
    l = tex1Dfetch(tex_out, left);
    c = tex1Dfetch(tex_out, offset);
    r = tex1Dfetch(tex_out, right);
    b = tex1Dfetch(tex_out, bottom);
  }

  dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

void anim_gpu(DataBlock *d, int ticks) {
  HANDLE_ERROR(hipEventRecord(d->start, 0));

  dim3 blocks(DIM / 16, DIM / 16);
  dim3 threads(16, 16);

  volatile bool dst_out = true;

  for (int i = 0; i < 90; ++i) {
    float *in, *out;

    if (dst_out) {
      in = d->dev_in_src;
      out = d->dev_out_src;
    } else {
      out = d->dev_in_src;
      in = d->dev_out_src;
    }

    copy_const_kernel<<<blocks, threads>>>(in);
    blend_kernel<<<blocks, threads>>>(out, dst_out);
    dst_out = !dst_out;
  }

  float_to_color<<<blocks, threads>>>(d->output_bitmap, d->dev_in_src);

  HANDLE_ERROR(hipMemcpy(d->bitmap->get_ptr(), d->output_bitmap,
  d->bitmap->image_size(), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipEventRecord(d->stop, 0));
  HANDLE_ERROR(hipEventSynchronize(d->stop));

  float elapsed_time;

  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time, d->start, d->stop));

  d->total_time += elapsed_time;
  ++d->frames;
  printf("Average time per frame: %3.1f ms\n", d->total_time / d->frames);
}

void anim_exit(DataBlock *d) {
  hipUnbindTexture(tex_in);
  hipUnbindTexture(tex_out);
  hipUnbindTexture(tex_const_src);

  hipFree(d->dev_in_src);
  hipFree(d->dev_out_src);
  hipFree(d->dev_const_src);

  HANDLE_ERROR(hipEventDestroy(d->start));
  HANDLE_ERROR(hipEventDestroy(d->stop));
}

int main(void) {
  DataBlock data;
  CPUAnimBitmap bitmap(DIM, DIM, &data);
  data.bitmap = &bitmap;
  data.total_time = 0;
  data.frames = 0;

  HANDLE_ERROR(hipEventCreate(&data.start));
  HANDLE_ERROR(hipEventCreate(&data.stop));

  HANDLE_ERROR(hipMalloc((void **)&data.output_bitmap, bitmap.image_size()));
  HANDLE_ERROR(hipMalloc((void **)&data.dev_in_src, bitmap.image_size()));
  HANDLE_ERROR(hipMalloc((void **)&data.dev_out_src, bitmap.image_size()));
  HANDLE_ERROR(hipMalloc((void **)&data.dev_const_src, bitmap.image_size()));

  HANDLE_ERROR(hipBindTexture(NULL, tex_const_src, data.dev_const_src,
                               bitmap.image_size()));
  HANDLE_ERROR(hipBindTexture(NULL, tex_in, data.dev_in_src,
                               bitmap.image_size()));
  HANDLE_ERROR(hipBindTexture(NULL, tex_out, data.dev_out_src,
                               bitmap.image_size()));

  float *temp = (float *)malloc(bitmap.image_size());

  for (int i = 0; i < DIM * DIM; ++i) {
    temp[i] = 0;

    int const x = i % DIM;
    int const y = i / DIM;

    if (x > 300 && x < 600 && y > 310 && y < 601) {
      temp[i] = MAX_TEMP;
    }
  }

  temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
  temp[DIM * 700 + 100] = MIN_TEMP;
  temp[DIM * 300 + 300] = MIN_TEMP;
  temp[DIM * 200 + 700] = MIN_TEMP;

  for (int y = 800; y < 900; ++y) {
    for (int x = 400; x < 500; ++x) {
      temp[x + y * DIM] = MIN_TEMP;
    }
  }

  HANDLE_ERROR(hipMemcpy(data.dev_const_src, temp, bitmap.image_size(),
                          hipMemcpyHostToDevice));

  for (int y = 800; y < DIM; ++y) {
    for (int x = 0; x < 200; ++x) {
      temp[x + y * DIM] = MAX_TEMP;
    }
  }

  HANDLE_ERROR(hipMemcpy(data.dev_in_src, temp, bitmap.image_size(),
                          hipMemcpyHostToDevice));

  free(temp);
  bitmap.anim_and_exit((void (*)(void *, int))anim_gpu,
                       (void (*)(void *))anim_exit);

  return 0;
}
