#include "hip/hip_runtime.h"
#include "common/book.h"

#define SIZE (100 * 1024 * 1024)

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {
  __shared__ unsigned int temp[256];
  temp[threadIdx.x] = 0;
  __syncthreads();

  int i      = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = blockDim.x * gridDim.x;
  while (i < size) {
    atomicAdd(&temp[buffer[i]], 1);
    i += offset;
  }

  __syncthreads();
  atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}

int main(int argc, char *argv[]) {
  unsigned char *buffer = (unsigned char*)big_random_block(SIZE);

  hipEvent_t start, stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  unsigned char *dev_buffer;
  unsigned int *dev_histo;

  HANDLE_ERROR(hipMalloc((void**)&dev_buffer, SIZE));
  HANDLE_ERROR(hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice));

  HANDLE_ERROR(hipMalloc((void**)&dev_histo, 256 * sizeof(long)));
  HANDLE_ERROR(hipMemset(dev_histo, 0, 256 * sizeof(int)));

  hipDeviceProp_t prop;
  HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
  int blocks = prop.multiProcessorCount;
  histo_kernel<<<blocks * 2, 256>>>(dev_buffer, SIZE, dev_histo);

  unsigned int histo[256];
  HANDLE_ERROR(hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));

  float elapsed_time;
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time, start, stop));
  printf("Time to generate: %3.1f ms\n", elapsed_time);

  long histo_count;
  for (int i = 0; i < 256; ++i) { histo_count += histo[i]; }
  printf("Histogram sum: %ld\n", histo_count);

  for (int i = 0; i < SIZE; ++i) { --histo[buffer[i]]; }
  for (int i = 0; i < 256; ++i) {
    if (histo[i] != 0) { printf("Failure at %d!\n", i); }
  }

  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));
  hipFree(dev_histo);
  hipFree(dev_buffer);
  free(buffer);

  return EXIT_SUCCESS;
}
