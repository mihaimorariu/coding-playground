
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 16;
const int block_size = 16;

__global__
void hello(char *a, int *b) {
  a[threadIdx.x] += b[threadIdx.x];
}

int main(int argc, char *argv[]) {
  char a[N] = "Hello ";
  int b[N] = {15, 10, 6, 0 -11, 1, 0};

  char *ad;
  int *bd;

  const int csize = N * sizeof(char);
  const int isize = N * sizeof(int);

  printf("%s", a);

  hipMalloc((void**)&ad, csize);
  hipMalloc((void**)&bd, isize);
  hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
  hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

  dim3 dim_block(block_size, 1);
  dim3 dim_grid(1, 1);
  hello<<<dim_grid, dim_block>>>(ad, bd);

  hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
  hipFree(ad);;
  hipFree(bd);

  printf("%s\n", a);

  return EXIT_SUCCESS;
}
