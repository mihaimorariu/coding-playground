#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/cpu_bitmap.h"

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#define DIM 1000

struct hipComplex {
  float r;
  float i;

  __device__ hipComplex(float a, float b) : r(a), i(b) {}
  __device__ float magnitude2(void) { return r * r + i * i; }

  __device__ hipComplex operator*(hipComplex const &a) {
    return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
  }

  __device__ hipComplex operator+(hipComplex const &a) {
    return hipComplex(r + a.r, i + a.i);
  }
};

__device__ int julia(int x, int y) {
  float const scale = 1.5;
  float const jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
  float const jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

  hipComplex c(-0.8, 0.156);
  hipComplex a(jx, jy);

  for (int i = 0; i < 200; ++i) {
    a = a * a + c;
    if (a.magnitude2() > 1000) {
      return 0;
    }
  }

  return 1;
}

__global__ void kernel(unsigned char *ptr) {
  int x = blockIdx.x;
  int y = blockIdx.y;
  int offset = x + y * gridDim.x;

  int julia_value = julia(x, y);
  ptr[offset * 4 + 0] = 255 * julia_value;
  ptr[offset * 4 + 1] = 255 * julia_value;
  ptr[offset * 4 + 2] = 0;
  ptr[offset * 4 + 3] = 255;
}

int main(void) {
  CPUBitmap bitmap(DIM, DIM);
  unsigned char *dev_bitmap;

  HANDLE_ERROR(hipMalloc((void **)&dev_bitmap, bitmap.image_size()));

  dim3 grid(DIM, DIM);
  kernel<<<grid, 1>>>(dev_bitmap);

  HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(),
                          hipMemcpyDeviceToHost));

  bitmap.display_and_exit();
  hipFree(dev_bitmap);

  return 0;
}
