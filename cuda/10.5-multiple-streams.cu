#include "hip/hip_runtime.h"
#include "common/book.h"

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)

__global__ void kernel(int *a, int *b, int *c) {
  int const idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < N) {
    int const idx1 = (idx + 1) % 256;
    int const idx2 = (idx + 2) % 256;

    float const as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
    float const bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;

    c[idx] = (as + bs) / 2;
  }
}

int main(void) {
  hipDeviceProp_t prop;
  int which_device;

  HANDLE_ERROR(hipGetDevice(&which_device));
  HANDLE_ERROR(hipGetDeviceProperties(&prop, which_device));

  if (!prop.deviceOverlap) {
    printf("Device %d will not handle overlaps, so no speed up from streams\n",
           which_device);
    return 0;
  }

  hipEvent_t start, stop;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  hipStream_t stream0, stream1;
  HANDLE_ERROR(hipStreamCreate(&stream0));
  HANDLE_ERROR(hipStreamCreate(&stream1));

  int *host_a, *host_b, *host_c;
  int *dev_a0, *dev_b0, *dev_c0;
  int *dev_a1, *dev_b1, *dev_c1;

  HANDLE_ERROR(hipMalloc((void **)&dev_a0, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b0, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_c0, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_a1, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b1, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_c1, N * sizeof(int)));

  HANDLE_ERROR(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int),
                             hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int),
                             hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int),
                             hipHostMallocDefault));

  for (int i = 0; i < FULL_DATA_SIZE; ++i) {
    host_a[i] = rand();
    host_b[i] = rand();
  }

  for (int i = 0; i < FULL_DATA_SIZE; i += N * 2) {
    HANDLE_ERROR(hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int),
                                 hipMemcpyHostToDevice, stream0));
    HANDLE_ERROR(hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int),
                                 hipMemcpyHostToDevice, stream0));
    kernel<<<N / 256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
    HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int),
                                 hipMemcpyDeviceToHost, stream0));
    HANDLE_ERROR(hipMemcpyAsync(dev_a1, host_a + i, N * sizeof(int),
                                 hipMemcpyHostToDevice, stream1));
    HANDLE_ERROR(hipMemcpyAsync(dev_b1, host_b + i, N * sizeof(int),
                                 hipMemcpyHostToDevice, stream1));
    kernel<<<N / 256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
    HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c1, N * sizeof(int),
                                 hipMemcpyDeviceToHost, stream1));
  }

  HANDLE_ERROR(hipStreamSynchronize(stream0));
  HANDLE_ERROR(hipStreamSynchronize(stream1));

  float elapsed_time;
  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time, start, stop));

  printf("Elapsed time: %3.1f ms\n", elapsed_time);

  HANDLE_ERROR(hipHostFree(host_a));
  HANDLE_ERROR(hipHostFree(host_b));
  HANDLE_ERROR(hipHostFree(host_c));

  HANDLE_ERROR(hipFree(dev_a0));
  HANDLE_ERROR(hipFree(dev_b0));
  HANDLE_ERROR(hipFree(dev_c0));
  HANDLE_ERROR(hipFree(dev_a1));
  HANDLE_ERROR(hipFree(dev_b1));
  HANDLE_ERROR(hipFree(dev_c1));

  HANDLE_ERROR(hipStreamDestroy(stream0));
  HANDLE_ERROR(hipStreamDestroy(stream1));

  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  return 0;
}
