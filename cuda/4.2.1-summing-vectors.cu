#include "hip/hip_runtime.h"
#include "common/book.h"

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

#define N 10000

__global__ void add(int *a, int *b, int *c) {
  int tid = blockIdx.x;
  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

int main(void) {
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  HANDLE_ERROR(hipMalloc((void **)&dev_a, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_c, N * sizeof(int)));

  for (int i = 0; i < N; ++i) {
    a[i] = -i;
    b[i] = i * i;
  }

  HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

  add<<<N, 1>>>(dev_a, dev_b, dev_c);

  HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

  for (int i = 0; i < N; ++i) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}
