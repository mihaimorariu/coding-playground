#include "hip/hip_runtime.h"
#include "common/book.h"

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

#define imin(a, b) (a < b ? a : b)
#define sum_squares(x) (x * (x + 1) * (2 * x + 1) / 6)

int const N = 33 * 1024;
int const threads_per_block = 256;
int const blocks_per_grid =
    imin(32, (N + threads_per_block - 1) / threads_per_block);

__global__ void dot(int size, float *a, float *b, float *c) {
  __shared__ float cache[threads_per_block];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int const cache_index = threadIdx.x;

  float temp = 0;
  while (tid < size) {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }

  cache[cache_index] = temp;
  __syncthreads();

  int i = blockDim.x / 2;
  while (i != 0) {
    if (cache_index < i) {
      cache[cache_index] += cache[cache_index + i];
      __syncthreads();
    }
    i /= 2;
  }

  if (cache_index == 0) {
    c[blockIdx.x] = cache[0];
  }
}

float malloc_test(int size) {
  hipEvent_t start, stop;
  float *a, *b, c, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;
  float elapsed_time;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  a = (float *)malloc(size * sizeof(float));
  b = (float *)malloc(size * sizeof(float));
  partial_c = (float *)malloc(blocks_per_grid * sizeof(float));

  HANDLE_ERROR(hipMalloc((void **)&dev_a, size * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b, size * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_partial_c, size * sizeof(float)));

  for (int i = 0; i < size; ++i) {
    a[i] = i;
    b[i] = i * 2;
  }

  HANDLE_ERROR(hipEventRecord(start, 0));
  HANDLE_ERROR(
      hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice));
  dot<<<blocks_per_grid, threads_per_block>>>(size, dev_a, dev_b,
                                              dev_partial_c);
  HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c,
                          blocks_per_grid * sizeof(float),
                          hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time, start, stop));

  c = 0;
  for (int i = 0; i < blocks_per_grid; ++i) {
    c += partial_c[i];
  }

  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_partial_c));

  free(a);
  free(b);
  free(partial_c);

  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  printf("Dot product: %f\n", c);

  return elapsed_time;
}

float cuda_host_malloc_test(int size) {
  hipEvent_t start, stop;
  float *a, *b, c, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;
  float elapsed_time;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipHostAlloc((void **)&a, size * sizeof(float),
                             hipHostMallocWriteCombined | hipHostMallocMapped));
  HANDLE_ERROR(hipHostAlloc((void **)&b, size * sizeof(float),
                             hipHostMallocWriteCombined | hipHostMallocMapped));
  HANDLE_ERROR(hipHostAlloc((void **)&partial_c, size * sizeof(float),
                             hipHostMallocMapped));

  for (int i = 0; i < size; ++i) {
    a[i] = i;
    b[i] = i * 2;
  }

  HANDLE_ERROR(hipHostGetDevicePointer(&dev_a, a, 0));
  HANDLE_ERROR(hipHostGetDevicePointer(&dev_b, b, 0));
  HANDLE_ERROR(hipHostGetDevicePointer(&dev_partial_c, partial_c, 0));

  HANDLE_ERROR(hipEventRecord(start, 0));
  dot<<<blocks_per_grid, threads_per_block>>>(size, dev_a, dev_b,
                                              dev_partial_c);
  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time, start, stop));

  c = 0;
  for (int i = 0; i < blocks_per_grid; ++i) {
    c += partial_c[i];
  }

  HANDLE_ERROR(hipHostFree(a));
  HANDLE_ERROR(hipHostFree(b));
  HANDLE_ERROR(hipHostFree(partial_c));

  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  printf("Dot product: %f\n", c);

  return elapsed_time;
}

int main(void) {
  hipDeviceProp_t prop;
  int which_device;

  HANDLE_ERROR(hipGetDevice(&which_device));
  HANDLE_ERROR(hipGetDeviceProperties(&prop, which_device));
  if (prop.canMapHostMemory != 1) {
    printf("Device %d cannot map memory.\n", which_device);
    return 0;
  }

  HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

  float elapsed_time = malloc_test(N);
  printf("Time using hipMalloc: %.5f ms\n", elapsed_time);

  elapsed_time = cuda_host_malloc_test(N);
  printf("Time using hipHostAlloc: %.5f ms\n", elapsed_time);

  return 0;
}
