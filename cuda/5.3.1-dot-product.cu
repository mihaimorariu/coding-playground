#include "hip/hip_runtime.h"
#include "common/book.h"

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

#define imin(a, b) (a < b ? a : b)
#define sum_squares(x) (x * (x + 1) * (2 * x + 1) / 6)

int const N = 33 * 1024;
int const threads_per_block = 256;
int const blocks_per_grid =
    imin(32, (N + threads_per_block - 1) / threads_per_block);

__global__ void dot(float *a, float *b, float *c) {
  __shared__ float cache[threads_per_block];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int const cache_index = threadIdx.x;

  float temp = 0;
  while (tid < N) {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }

  cache[cache_index] = temp;
  __syncthreads();

  int i = blockDim.x / 2;
  while (i != 0) {
    if (cache_index < i) {
      cache[cache_index] += cache[cache_index + i];
      __syncthreads();
    }
    i /= 2;
  }

  if (cache_index == 0) {
    c[blockIdx.x] = cache[0];
  }
}

int main(void) {
  float *a, *b, c, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;

  a = (float*)malloc(N * sizeof(float));
  b = (float*)malloc(N * sizeof(float));
  partial_c = (float*)malloc(blocks_per_grid * sizeof(float));

  HANDLE_ERROR(hipMalloc((void **)&dev_a, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b, N * sizeof(float)));
  HANDLE_ERROR(
      hipMalloc((void **)&dev_partial_c, blocks_per_grid * sizeof(float)));

  for (int i = 0; i < N; ++i) {
    a[i] = i;
    b[i] = i * 2;
  }

  HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));

  dot<<<blocks_per_grid, threads_per_block>>>(dev_a, dev_b, dev_partial_c);

  HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c,
                          blocks_per_grid * sizeof(float),
                          hipMemcpyDeviceToHost));

  c = 0;
  for (int i = 0; i < blocks_per_grid; ++i) {
    c += partial_c[i];
  }

  printf("real dot product = %.6g\n", 2 * sum_squares((float)(N  - 1)));
  printf("computed dot product = %.6g\n", c);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_partial_c);

  delete[] a;
  delete[] b;
  delete[] partial_c;

  return 0;
}
